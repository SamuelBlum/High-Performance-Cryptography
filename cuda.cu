#include "hip/hip_runtime.h"
#include <iostream>
#include <string>

using namespace std;

__global__ void convertData(char* text, int length)
{
    for (unsigned int i = 0; i < length; i++)
    {
        //ALPHA
        if (text[i] >= 'A')
    }
}

int main()
{

    char input[] = "aabfdh.fdsjkl.+@!ckdsj/khj";

    cout << "INPUT:  " << input << endl;

    int N = sizeof(input);

    char* d_text;

    //Memory Allocation

    hipMalloc(&d_text, N*sizeof(char));

    //Copy input datastream to GPU

    hipMemcpy(d_text, input, N*sizeof(char), hipMemcpyHostToDevice);

    convertData <<<1, 1>>>(d_text, N);

    //Copy output datastream back to the CPU

    hipMemcpy(input, d_text, N*sizeof(char), hipMemcpyDeviceToHost);

    cout << "OUTPUT: " << input << endl;

    hipFree(d_text);

    return 0;

}
