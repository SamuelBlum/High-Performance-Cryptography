
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

using namespace std;

__global__ void convertData(char* text, int length)
{
    for (unsigned int i = 0; i < length; i++)
    {
        //ALPHA
        if (text[i] == 'a')
            text[i] = 'A';
        else if (text[i] == 'b')
            text[i] = 'B';
        else if (text[i] == 'c')
            text[i] = 'C';
        else if (text[i] == 'd')
            text[i] = 'D';
        else if (text[i] == 'e')
            text[i] = 'E';
        else if (text[i] == 'f')
            text[i] = 'F';
        else if (text[i] == 'g')
            text[i] = 'G';
        else if (text[i] == 'h')
            text[i] = 'H';
        else if (text[i] == 'i')
            text[i] = 'I';
        else if (text[i] == 'j')
            text[i] = 'J';
        else if (text[i] == 'k')
            text[i] = 'K';
        else if (text[i] == 'l')
            text[i] = 'L';
        else if (text[i] == 'm')
            text[i] = 'M';
        else if (text[i] == 'n')
            text[i] = 'N';
        else if (text[i] == 'o')
            text[i] = 'O';
        else if (text[i] == 'p')
            text[i] = 'P';
        else if (text[i] == 'q')
            text[i] = 'Q';
        else if (text[i] == 'r')
            text[i] = 'R';
        else if (text[i] == 's')
            text[i] = 'S';
        else if (text[i] == 't')
            text[i] = 'T';
        else if (text[i] == 'u')
            text[i] = 'U';
        else if (text[i] == 'v')
            text[i] = 'V';
        else if (text[i] == 'w')
            text[i] = 'W';
        else if (text[i] == 'x')
            text[i] = 'X';
        else if (text[i] == 'y')
            text[i] = 'Y';
        else if (text[i] == 'z')
            text[i] = 'Z';
        //NON-ALPHANUMERIC
        else if ((text[i] == '/'))
            text[i] = '.';
        else if ((text[i] == '?'))
            text[i] = '.';
        else if ((text[i] == ','))
            text[i] = '.';
        else if ((text[i] == '"'))
            text[i] = '.';
        else if ((text[i] == '`'))
            text[i] = '.';
        else if ((text[i] == '!'))
            text[i] = '.';
        else if ((text[i] == '@'))
            text[i] = '.';
        else if ((text[i] == '#'))
            text[i] = '.';
        else if ((text[i] == '$'))
            text[i] = '.';
        else if ((text[i] == '%'))
            text[i] = '.';
        else if ((text[i] == '^'))
            text[i] = '.';
        else if ((text[i] == '&'))
            text[i] = '.';
        else if ((text[i] == '*'))
            text[i] = '.';
        else if ((text[i] == '{'))
            text[i] = '.';
        else if ((text[i] == '}'))
            text[i] = '.';
        else if ((text[i] == '|'))
            text[i] = '.';
        else if ((text[i] == '('))
            text[i] = '.';        
        else if ((text[i] == '('))
            text[i] = '.';
        else if ((text[i] == ' '))
            text[i] = '.';
        else if ((text[i] == '+'))
            text[i] = '.';
        else if((text[i] == '='))
            text[i] = '.';
    }
}

int main()
{

    char input[] = "aabfdh.fdsjkl.+@!ckdsj/khj";

    cout << "INPUT:  " << input << endl;

    int N = sizeof(input);

    char* d_text;

    hipMalloc(&d_text, N*sizeof(char));

    hipMemcpy(d_text, input, N*sizeof(char), hipMemcpyHostToDevice);

    convertData <<<1, 1>>>(d_text, N);

    hipMemcpy(input, d_text, N*sizeof(char), hipMemcpyDeviceToHost);

    cout << "OUTPUT: " << input << endl;

    hipFree(d_text);

    return 0;

}
